#include "hip/hip_runtime.h"
#include <exchcxx/xc_functional.hpp>


__global__ void scal_kernel( const int N, const double fact, const double* X_device, double* Y_device ) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if( tid < N ) Y_device[tid] = X_device[tid] * fact;

}

__global__ void add_scal_kernel( const int N, const double fact, const double* X_device, double* Y_device ) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if( tid < N ) Y_device[tid] += X_device[tid] * fact;

}

void scal_device( const int N, const double fact, const double* X_device, double* Y_device ) {
  int threads = 1024;
  int blocks  = std::ceil( N / 1024. );
  scal_kernel<<< blocks, threads >>>( N, fact, X_device, Y_device );
}

void scal_device( const int N, const double fact, const double* X_device, double* Y_device, hipStream_t& stream ) {
  int threads = 1024;
  int blocks  = std::ceil( N / 1024. );
  scal_kernel<<< blocks, threads, 0, stream >>>( N, fact, X_device, Y_device );
}

void add_scal_device( const int N, const double fact, const double* X_device, double* Y_device ) {
  int threads = 1024;
  int blocks  = std::ceil( N / 1024. );
  add_scal_kernel<<< blocks, threads >>>( N, fact, X_device, Y_device );
}

void add_scal_device( const int N, const double fact, const double* X_device, double* Y_device, hipStream_t& stream ) {
  int threads = 1024;
  int blocks  = std::ceil( N / 1024. );
  add_scal_kernel<<< blocks, threads, 0, stream >>>( N, fact, X_device, Y_device );
}


template <typename T = double>
T* safe_cuda_malloc( size_t N ) {

  T* ptr = nullptr;
  auto stat = hipMalloc( &ptr, N*sizeof(T) );
  if( stat != hipSuccess ) throw std::runtime_error("Alloc Failed");

  return ptr;

}

namespace ExchCXX {






LDA_EXC_GENERATOR_DEVICE( XCFunctional::eval_exc_device ) const {

  throw_if_not_sane();
  assert( is_lda() );

  double* eps_scr = nullptr;
  if( kernels_.size() > 1 ) 
    eps_scr = safe_cuda_malloc( N );

  for( auto i = 0ul; i < kernels_.size(); ++i ) {

    double* eps_eval = i ? eps_scr : eps;
    kernels_[i].second.eval_exc_device(N, rho, eps_eval, stream);

    if( i ) 
      add_scal_device( N, kernels_[i].first, eps_eval, eps, stream );
    else
      scal_device( N, kernels_[i].first, eps_eval, eps, stream );
  
  }

  if( eps_scr ) hipFree( eps_scr );

}


LDA_EXC_VXC_GENERATOR_DEVICE( XCFunctional::eval_exc_vxc_device ) const {

  throw_if_not_sane();
  assert( is_lda() );

  int len_vxc = is_polarized() ? 2*N : N;

  double* eps_scr(nullptr), *vxc_scr(nullptr);
  if( kernels_.size() > 1 ) {
    eps_scr = safe_cuda_malloc( N );
    vxc_scr = safe_cuda_malloc( len_vxc );
  }

  for( auto i = 0ul; i < kernels_.size(); ++i ) {

    double* eps_eval = i ? eps_scr : eps;
    double* vxc_eval = i ? vxc_scr : vxc;
    kernels_[i].second.eval_exc_vxc_device(N, rho, eps_eval, vxc_eval, stream);

    if( i ) {

      add_scal_device( N,       kernels_[i].first, eps_eval, eps, stream );
      add_scal_device( len_vxc, kernels_[i].first, vxc_eval, vxc, stream );

    } else {

      scal_device( N,       kernels_[i].first, eps_eval, eps, stream );
      scal_device( len_vxc, kernels_[i].first, vxc_eval, vxc, stream );

    }
  
  }

  if( eps_scr ) hipFree( eps_scr );
  if( vxc_scr ) hipFree( vxc_scr );

}



// GGA Interfaces

GGA_EXC_GENERATOR_DEVICE( XCFunctional::eval_exc_device ) const {

  throw_if_not_sane();
  assert( is_gga() );

  double* eps_scr = nullptr;
  if( kernels_.size() > 1 ) 
    eps_scr = safe_cuda_malloc( N );


  for( auto i = 0ul; i < kernels_.size(); ++i ) {

    double* eps_eval = i ? eps_scr : eps;

    if( kernels_[i].second.is_gga() )
      kernels_[i].second.eval_exc_device(N, rho, sigma, eps_eval, stream);
    else
      kernels_[i].second.eval_exc_device(N, rho, eps_eval, stream);

    if( i ) 
      add_scal_device( N, kernels_[i].first, eps_eval, eps, stream );
    else
      scal_device( N, kernels_[i].first, eps_eval, eps, stream );
  
  }

  if( eps_scr ) hipFree( eps_scr );

}


GGA_EXC_VXC_GENERATOR_DEVICE( XCFunctional::eval_exc_vxc_device ) const {

  throw_if_not_sane();
  assert( is_gga() );

  int len_vrho   = is_polarized() ? 2*N : N;
  int len_vsigma = is_polarized() ? 3*N : N;

  double* eps_scr(nullptr), *vrho_scr(nullptr), *vsigma_scr(nullptr);
  if( kernels_.size() > 1 ) {
    eps_scr    = safe_cuda_malloc( N );
    vrho_scr   = safe_cuda_malloc( len_vrho );
    vsigma_scr = safe_cuda_malloc( len_vsigma );
  }

  for( auto i = 0ul; i < kernels_.size(); ++i ) {

    double* eps_eval    = i ? eps_scr    : eps;
    double* vrho_eval   = i ? vrho_scr   : vrho;
    double* vsigma_eval = i ? vsigma_scr : vsigma;

    if( kernels_[i].second.is_gga() )
      kernels_[i].second.eval_exc_vxc_device(N, rho, sigma, eps_eval, vrho_eval, 
        vsigma_eval, stream );
    else
      kernels_[i].second.eval_exc_vxc_device(N, rho, eps_eval, vrho_eval, stream);

    if( i ) {

      add_scal_device( N, kernels_[i].first, eps_eval, eps, stream );
      add_scal_device( len_vrho, kernels_[i].first, vrho_eval, vrho, stream);
      if( kernels_[i].second.is_gga() )
        add_scal_device( len_vsigma, kernels_[i].first, vsigma_eval, vsigma, stream );

    } else {

      scal_device( N, kernels_[i].first, eps_eval, eps, stream );
      scal_device( len_vrho, kernels_[i].first, vrho_eval, vrho, stream );
      if( kernels_[i].second.is_gga() )
        scal_device( len_vsigma, kernels_[i].first, vsigma_eval, vsigma, stream );

    }
  
  }

  if( eps_scr )    hipFree( eps_scr );
  if( vrho_scr )   hipFree( vrho_scr );
  if( vsigma_scr ) hipFree( vsigma_scr );

}




// mGGA Interfaces

MGGA_EXC_GENERATOR_DEVICE( XCFunctional::eval_exc_device ) const {

  throw_if_not_sane();
  assert( is_mgga() );

  double* eps_scr = nullptr;
  if( kernels_.size() > 1 ) 
    eps_scr = safe_cuda_malloc( N );


  for( auto i = 0ul; i < kernels_.size(); ++i ) {

    double* eps_eval = i ? eps_scr : eps;

    if( kernels_[i].second.is_mgga() )
      kernels_[i].second.eval_exc_device(N, rho, sigma, lapl, tau, eps_eval, stream);
    else if( kernels_[i].second.is_gga() )
      kernels_[i].second.eval_exc_device(N, rho, sigma, eps_eval, stream);
    else
      kernels_[i].second.eval_exc_device(N, rho, eps_eval, stream);

    if( i ) 
      add_scal_device( N, kernels_[i].first, eps_eval, eps, stream );
    else
      scal_device( N, kernels_[i].first, eps_eval, eps, stream );
  
  }

  if( eps_scr ) hipFree( eps_scr );

}


MGGA_EXC_VXC_GENERATOR_DEVICE( XCFunctional::eval_exc_vxc_device ) const {

  throw_if_not_sane();
  assert( is_gga() );

  int len_vrho   = is_polarized() ? 2*N : N;
  int len_vsigma = is_polarized() ? 3*N : N;
  int len_vlapl  = is_polarized() ? 2*N : N;
  int len_vtau   = is_polarized() ? 2*N : N;

  double* eps_scr(nullptr), *vrho_scr(nullptr), *vsigma_scr(nullptr), 
    *vlapl_scr(nullptr), *vtau_scr(nullptr);
  if( kernels_.size() > 1 ) {
    eps_scr    = safe_cuda_malloc( N );
    vrho_scr   = safe_cuda_malloc( len_vrho );
    vsigma_scr = safe_cuda_malloc( len_vsigma );
    vlapl_scr  = safe_cuda_malloc( len_vlapl );
    vtau_scr   = safe_cuda_malloc( len_vtau );
  }

  for( auto i = 0ul; i < kernels_.size(); ++i ) {

    double* eps_eval    = i ? eps_scr    : eps;
    double* vrho_eval   = i ? vrho_scr   : vrho;
    double* vsigma_eval = i ? vsigma_scr : vsigma;
    double* vlapl_eval  = i ? vlapl_scr  : vlapl;
    double* vtau_eval   = i ? vtau_scr   : vtau;

    if( kernels_[i].second.is_mgga() )
      kernels_[i].second.eval_exc_vxc_device(N, rho, sigma, lapl, tau, eps_eval, 
        vrho_eval, vsigma_eval, vlapl_eval, vtau_eval, stream );
    else if( kernels_[i].second.is_gga() )
      kernels_[i].second.eval_exc_vxc_device(N, rho, sigma, eps_eval, vrho_eval, 
        vsigma_eval, stream );
    else
      kernels_[i].second.eval_exc_vxc_device(N, rho, eps_eval, vrho_eval, stream);

    if( i ) {

      add_scal_device( N, kernels_[i].first, eps_eval, eps, stream );
      add_scal_device( len_vrho, kernels_[i].first, vrho_eval, vrho, stream );

      if( kernels_[i].second.is_gga() )
        add_scal_device( len_vsigma, kernels_[i].first, vsigma_eval, vsigma, stream );

      if( kernels_[i].second.is_mgga() ) {
        add_scal_device( len_vlapl, kernels_[i].first, vlapl_eval, vlapl, stream );
        add_scal_device( len_vtau,  kernels_[i].first, vtau_eval,  vtau, stream  );
      }

    } else {

      scal_device( N, kernels_[i].first, eps_eval, eps, stream );
      scal_device( len_vrho, kernels_[i].first, vrho_eval, vrho, stream );

      if( kernels_[i].second.is_gga() )
        scal_device( len_vsigma, kernels_[i].first, vsigma_eval, vsigma, stream );

      if( kernels_[i].second.is_mgga() ) {
        scal_device( len_vlapl, kernels_[i].first, vlapl_eval, vlapl, stream );
        scal_device( len_vtau,  kernels_[i].first, vtau_eval,  vtau, stream  );
      }

    }
  
  }

  if( eps_scr )    hipFree( eps_scr );
  if( vrho_scr )   hipFree( vrho_scr );
  if( vsigma_scr ) hipFree( vsigma_scr );
  if( vlapl_scr )  hipFree( vlapl_scr );
  if( vtau_scr )   hipFree( vtau_scr );
}

}
