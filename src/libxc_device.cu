#include "hip/hip_runtime.h"
#include "libxc_common.hpp"
#include <exchcxx/device/cuda_type_wrappers.hpp>

//#include <functionals.cuh>

void throw_if_fail( hipError_t stat, std::string msg ) {
  if( stat != hipSuccess ) throw std::runtime_error( msg );
}

void recv_from_device( void* dest, const void* src, const size_t len ) {

  auto stat = hipMemcpy( dest, src, len, hipMemcpyDeviceToHost );
  throw_if_fail( stat, "recv failed" );

}

void recv_from_device( void* dest, const void* src, const size_t len, 
  hipStream_t& stream ) {

  auto stat = hipMemcpyAsync( dest, src, len, hipMemcpyDeviceToHost, stream );
  throw_if_fail( stat, "recv failed" );

}

void send_to_device( void* dest, const void* src, const size_t len ) {

  auto stat = hipMemcpy( dest, src, len, hipMemcpyHostToDevice);
  throw_if_fail( stat, "send failed" );

}

void send_to_device( void* dest, const void* src, const size_t len, 
  hipStream_t& stream ) {

  auto stat = hipMemcpyAsync( dest, src, len, hipMemcpyHostToDevice, stream);
  throw_if_fail( stat, "send failed" );

}

void stream_sync( hipStream_t& stream ) {

  auto stat = hipStreamSynchronize( stream );
  throw_if_fail( stat, "sync failed" );

}

namespace ExchCXX {

namespace detail {

// LDA interfaces
LDA_EXC_GENERATOR( LibxcKernelImpl::eval_exc_device_ ) const {

  throw_if_uninitialized();
  assert( is_lda() );

  size_t len_rho = N*sizeof(double);
  size_t len_eps = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N );

  recv_from_device( rho_host.data(), rho, len_rho );

  xc_lda_exc( &kernel_, N, rho_host.data(), eps_host.data() );

  send_to_device( eps, eps_host.data(), len_eps );

}


LDA_EXC_VXC_GENERATOR( LibxcKernelImpl::eval_exc_vxc_device_ ) const {

  throw_if_uninitialized();
  assert( is_lda() );

  size_t len_rho = N*sizeof(double);
  size_t len_eps = N*sizeof(double);
  size_t len_vxc = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N ), vxc_host( N );

  recv_from_device( rho_host.data(), rho, len_rho );

  xc_lda_exc_vxc( &kernel_, N, rho_host.data(), eps_host.data(), vxc_host.data() );

  send_to_device( eps, eps_host.data(), len_eps );
  send_to_device( vxc, vxc_host.data(), len_vxc );

}

// TODO: LDA kxc interfaces

// GGA interface
GGA_EXC_GENERATOR( LibxcKernelImpl::eval_exc_device_ ) const {

  throw_if_uninitialized();
  assert( is_gga() );

  size_t len_rho   = N*sizeof(double);
  size_t len_eps   = N*sizeof(double);
  size_t len_sigma = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N ), sigma_host( N );

  recv_from_device( rho_host.data(),   rho,   len_rho   );
  recv_from_device( sigma_host.data(), sigma, len_sigma );

  xc_gga_exc( &kernel_, N, rho_host.data(), sigma_host.data(), eps_host.data() );

  send_to_device( eps, eps_host.data(), len_eps );

}


GGA_EXC_VXC_GENERATOR( LibxcKernelImpl::eval_exc_vxc_device_ ) const {

  throw_if_uninitialized();
  assert( is_gga() );


  size_t len_rho    = N*sizeof(double);
  size_t len_sigma  = N*sizeof(double);
  size_t len_vrho   = N*sizeof(double);
  size_t len_vsigma = N*sizeof(double);
  size_t len_eps    = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N ), sigma_host( N ), vrho_host( N ), 
                      vsigma_host( N );

  recv_from_device( rho_host.data(),   rho,   len_rho   );
  recv_from_device( sigma_host.data(), sigma, len_sigma );

  xc_gga_exc_vxc( &kernel_, N, rho_host.data(), sigma_host.data(), eps_host.data(), 
                  vrho_host.data(), vsigma_host.data() );

  send_to_device( eps,    eps_host.data(),    len_eps    );
  send_to_device( vrho,   vrho_host.data(),   len_vrho   );
  send_to_device( vsigma, vsigma_host.data(), len_vsigma );


}

// TODO: GGA kxc interfaces  
  
  
// mGGA interface
MGGA_EXC_GENERATOR( LibxcKernelImpl::eval_exc_device_ ) const {

  throw_if_uninitialized();
  assert( is_mgga() );

  size_t len_rho   = N*sizeof(double);
  size_t len_sigma = N*sizeof(double);
  size_t len_lapl  = N*sizeof(double);
  size_t len_tau   = N*sizeof(double);
  size_t len_eps   = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N ), sigma_host( N ), lapl_host( N ), 
                      tau_host( N );

  recv_from_device( rho_host.data(),   rho,   len_rho   );
  recv_from_device( sigma_host.data(), sigma, len_sigma );
  recv_from_device( lapl_host.data(),  lapl,  len_lapl  );
  recv_from_device( tau_host.data(),   tau,   len_tau   );

  xc_mgga_exc( &kernel_, N, rho_host.data(), sigma_host.data(), lapl_host.data(), 
               tau_host.data(), eps_host.data() );

  send_to_device( eps, eps_host.data(), len_eps );

}


MGGA_EXC_VXC_GENERATOR( LibxcKernelImpl::eval_exc_vxc_device_ ) const {

  throw_if_uninitialized();
  assert( is_mgga() );

  size_t len_rho    = N*sizeof(double);
  size_t len_sigma  = N*sizeof(double);
  size_t len_lapl   = N*sizeof(double);
  size_t len_tau    = N*sizeof(double);
  size_t len_eps    = N*sizeof(double);
  size_t len_vrho   = N*sizeof(double);
  size_t len_vsigma = N*sizeof(double);
  size_t len_vlapl  = N*sizeof(double);
  size_t len_vtau   = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N ), sigma_host( N ), lapl_host( N ),                       tau_host( N );
  std::vector<double> vrho_host( N ), vsigma_host( N ),  vlapl_host( N ), 
                      vtau_host( N );

  recv_from_device( rho_host.data(),   rho,   len_rho   );
  recv_from_device( sigma_host.data(), sigma, len_sigma );
  recv_from_device( lapl_host.data(),  lapl,  len_lapl  );
  recv_from_device( tau_host.data(),   tau,   len_tau   );

  xc_mgga_exc_vxc( &kernel_, N, rho_host.data(), sigma_host.data(), 
                   lapl_host.data(), tau_host.data(), eps_host.data(), 
                   vrho_host.data(), vsigma_host.data(), vlapl_host.data(), 
                   vtau_host.data() );

  send_to_device( eps, eps_host.data(), len_eps );
  send_to_device( vrho,   vrho_host.data(),   len_vrho   );
  send_to_device( vsigma, vsigma_host.data(), len_vsigma );
  send_to_device( vlapl,  vlapl_host.data(),  len_vlapl  );
  send_to_device( vtau,   vtau_host.data(),   len_vtau   );
}






// LDA interfaces
LDA_EXC_GENERATOR_DEVICE( LibxcKernelImpl::eval_exc_device_async_ ) const {

  throw_if_uninitialized();
  assert( is_lda() );

  size_t len_rho = N*sizeof(double);
  size_t len_eps = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N );

  recv_from_device( rho_host.data(), rho, len_rho );

  xc_lda_exc( &kernel_, N, rho_host.data(), eps_host.data() );

  send_to_device( eps, eps_host.data(), len_eps );

}


LDA_EXC_VXC_GENERATOR_DEVICE( LibxcKernelImpl::eval_exc_vxc_device_async_ ) const {

  throw_if_uninitialized();
  assert( is_lda() );

  size_t len_rho = N*sizeof(double);
  size_t len_eps = N*sizeof(double);
  size_t len_vxc = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N ), vxc_host( N );

  recv_from_device( rho_host.data(), rho, len_rho );

  xc_lda_exc_vxc( &kernel_, N, rho_host.data(), eps_host.data(), vxc_host.data() );

  send_to_device( eps, eps_host.data(), len_eps );
  send_to_device( vxc, vxc_host.data(), len_vxc );

}

// TODO: LDA kxc interfaces

// GGA interface
GGA_EXC_GENERATOR_DEVICE( LibxcKernelImpl::eval_exc_device_async_ ) const {

  throw_if_uninitialized();
  assert( is_gga() );

  size_t len_rho   = N*sizeof(double);
  size_t len_eps   = N*sizeof(double);
  size_t len_sigma = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N ), sigma_host( N );

  recv_from_device( rho_host.data(),   rho,   len_rho   );
  recv_from_device( sigma_host.data(), sigma, len_sigma );

  xc_gga_exc( &kernel_, N, rho_host.data(), sigma_host.data(), eps_host.data() );

  send_to_device( eps, eps_host.data(), len_eps );

}


GGA_EXC_VXC_GENERATOR_DEVICE( LibxcKernelImpl::eval_exc_vxc_device_async_ ) const {

  throw_if_uninitialized();
  assert( is_gga() );

#if 1

  size_t len_rho    = N*sizeof(double);
  size_t len_sigma  = N*sizeof(double);
  size_t len_vrho   = N*sizeof(double);
  size_t len_vsigma = N*sizeof(double);
  size_t len_eps    = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N ), sigma_host( N ), vrho_host( N ), vsigma_host( N );

  hipStream_t& st = *stream->stream;
  recv_from_device( rho_host.data(),   rho,   len_rho  , st );
  recv_from_device( sigma_host.data(), sigma, len_sigma, st );
 
  stream_sync( st );
  xc_gga_exc_vxc( &kernel_, N, rho_host.data(), sigma_host.data(), eps_host.data(), vrho_host.data(), vsigma_host.data() );

  send_to_device( eps,    eps_host.data(),    len_eps    );
  send_to_device( vrho,   vrho_host.data(),   len_vrho   );
  send_to_device( vsigma, vsigma_host.data(), len_vsigma );

#else

  //dim3 threads = 1024;
  //dim3 blocks  = std::ceil( N / 1024. );
  //xc_gga_exc_vxc_device<<< blocks, threads >>>( &kernel_, N, rho_device, 
  //  sigma_device, eps_device, vrho_device, vsigma_device );

  hipError_t stat;
  stat = hipMemsetAsync( eps_device,    0, N*sizeof(double), *stream->stream ); throw_if_fail( stat, "EPS    ZERO" );
  stat = hipMemsetAsync( vrho_device,   0, N*sizeof(double), *stream->stream ); throw_if_fail( stat, "VRHO   ZERO" );
  stat = hipMemsetAsync( vsigma_device, 0, N*sizeof(double), *stream->stream ); throw_if_fail( stat, "VSIGMA ZERO" );

#endif

}

// TODO: GGA kxc interfaces  
  
  
// mGGA interface
MGGA_EXC_GENERATOR_DEVICE( LibxcKernelImpl::eval_exc_device_async_ ) const {

  throw_if_uninitialized();
  assert( is_mgga() );

  size_t len_rho   = N*sizeof(double);
  size_t len_sigma = N*sizeof(double);
  size_t len_lapl  = N*sizeof(double);
  size_t len_tau   = N*sizeof(double);
  size_t len_eps   = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N ), sigma_host( N ), lapl_host( N ), tau_host( N );

  recv_from_device( rho_host.data(),   rho,   len_rho   );
  recv_from_device( sigma_host.data(), sigma, len_sigma );
  recv_from_device( lapl_host.data(),  lapl,  len_lapl  );
  recv_from_device( tau_host.data(),   tau,   len_tau   );

  xc_mgga_exc( &kernel_, N, rho_host.data(), sigma_host.data(), lapl_host.data(), tau_host.data(), eps_host.data() );

  send_to_device( eps, eps_host.data(), len_eps );

}


MGGA_EXC_VXC_GENERATOR_DEVICE( LibxcKernelImpl::eval_exc_vxc_device_async_ ) const {

  throw_if_uninitialized();
  assert( is_mgga() );

  size_t len_rho    = N*sizeof(double);
  size_t len_sigma  = N*sizeof(double);
  size_t len_lapl   = N*sizeof(double);
  size_t len_tau    = N*sizeof(double);
  size_t len_eps    = N*sizeof(double);
  size_t len_vrho   = N*sizeof(double);
  size_t len_vsigma = N*sizeof(double);
  size_t len_vlapl  = N*sizeof(double);
  size_t len_vtau   = N*sizeof(double);

  std::vector<double> rho_host( N ), eps_host( N ), sigma_host( N ), lapl_host( N ), tau_host( N );
  std::vector<double> vrho_host( N ), vsigma_host( N ),  vlapl_host( N ), vtau_host( N );

  recv_from_device( rho_host.data(),   rho,   len_rho   );
  recv_from_device( sigma_host.data(), sigma, len_sigma );
  recv_from_device( lapl_host.data(),  lapl,  len_lapl  );
  recv_from_device( tau_host.data(),   tau,   len_tau   );

  xc_mgga_exc_vxc( &kernel_, N, rho_host.data(), sigma_host.data(), lapl_host.data(), tau_host.data(), eps_host.data(), vrho_host.data(), vsigma_host.data(), vlapl_host.data(), vtau_host.data() );

  send_to_device( eps, eps_host.data(), len_eps );
  send_to_device( vrho,   vrho_host.data(),   len_vrho   );
  send_to_device( vsigma, vsigma_host.data(), len_vsigma );
  send_to_device( vlapl,  vlapl_host.data(),  len_vlapl  );
  send_to_device( vtau,   vtau_host.data(),   len_vtau   );
}

}
}
